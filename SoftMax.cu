#include "hip/hip_runtime.h"
#include <THC/THC.h>

#define MINUS_LOG_THRESHOLD -18.42
#define SOFTMAX_THREADS 128

__global__ void cunn_SoftMax_updateOutput_kernel(float *output, float *input, int nframe, int dim)
{
  __shared__ float buffer[SOFTMAX_THREADS+1];
  int k = blockIdx.x;
  float *input_k = input + k*dim;
  float *output_k = output + k*dim;

  int i_start = threadIdx.x;
  int i_end = dim;
  int i_step = blockDim.x;

  // max?
  buffer[threadIdx.x] = -FLT_MAX;
  for (int i=i_start; i<i_end; i+=i_step)
  {
    float z = input_k[i];
    if(buffer[threadIdx.x] < z)
      buffer[threadIdx.x] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float max_k = -FLT_MAX;
    for (int i=0; i<blockDim.x; i++)
    {
      if(max_k < buffer[i])
        max_k = buffer[i];
    }
    buffer[SOFTMAX_THREADS] = max_k;
  }

  __syncthreads();

  // sum?
  float max_k = buffer[SOFTMAX_THREADS];
  buffer[threadIdx.x] = 0;
  for (int i=i_start; i<i_end; i+=i_step) {
    float z = __expf(input_k[i]-max_k);
    buffer[threadIdx.x] += z;
    output_k[i] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float sum_k = 0;
    for (int i=0; i<blockDim.x; i++)
      sum_k += buffer[i];
    buffer[SOFTMAX_THREADS] = sum_k;
  }

  __syncthreads();

  // softmax
  float sum_k = buffer[SOFTMAX_THREADS];
  for (int i=i_start; i<i_end; i+=i_step)
    output_k[i] = output_k[i] / sum_k;
}

extern "C"
void cunnrelease_SoftMax_updateOutput(THCState *state,
    THCudaTensor *input, THCudaTensor *output)
{
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));

  input = THCudaTensor_newContiguous(state, input);
  THCudaTensor_resizeAs(state, output, input);

  if(input->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(SOFTMAX_THREADS);
    cunn_SoftMax_updateOutput_kernel<<<blocks,threads,
      0, THCState_getCurrentStream(state)>>>(THCudaTensor_data(state, output),
                                             THCudaTensor_data(state, input),
                                             1, input->size[0]);
  }
  else if(input->nDimension == 2)
  {
    dim3 blocks(input->size[0]);
    dim3 threads(SOFTMAX_THREADS);
    cunn_SoftMax_updateOutput_kernel<<<blocks,threads,
      0, THCState_getCurrentStream(state)>>>(THCudaTensor_data(state, output),
                                             THCudaTensor_data(state, input),
                                             input->size[0], input->size[1]);
  }
  else
    THError("vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(state, input);
}
